#include "hip/hip_runtime.h"
#include "Atom_ExchangeCUDA.h"

#if COMPILECUDA == 1

#if defined(MODULE_COMPILATION_EXCHANGE) && ATOMISTIC == 1

#include "BorisCUDALib.cuh"

#include "Atom_MeshCUDA.h"
#include "Atom_MeshParamsControlCUDA.h"
#include "MeshDefs.h"

//////////////////////////////////////////////////////////////////////// UPDATE FIELD

__global__ void Atom_ExchangeCUDA_Cubic_UpdateField(ManagedAtom_MeshCUDA& cuaMesh, ManagedModulesCUDA& cuModule, bool do_reduction)
{
	cuVEC_VC<cuReal3>& M1 = *cuaMesh.pM1;
	cuVEC<cuReal3>& Heff1 = *cuaMesh.pHeff1;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal energy_ = 0.0;

	if (idx < Heff1.linear_size()) {

		cuReal3 Hexch = cuReal3();

		if (M1.is_not_empty(idx)) {

			cuBReal mu_s = *cuaMesh.pmu_s;
			cuBReal J = *cuaMesh.pJ;
			cuaMesh.update_parameters_mcoarse(idx, *cuaMesh.pmu_s, mu_s, *cuaMesh.pJ, J);

			//update effective field with the Heisenberg exchange field
			Hexch = (J / (MUB_MU0*mu_s)) * M1.ngbr_dirsum(idx);

			if (do_reduction) {

				//energy E = -mu_s * Bex
				//update energy density
				cuBReal non_empty_volume = M1.get_nonempty_cells() * M1.h.dim();
				if (non_empty_volume) energy_ = -(cuBReal)MUB_MU0 * M1[idx] * Hexch / (2*non_empty_volume);
			}

			if (do_reduction && cuModule.pModule_Heff->linear_size()) (*cuModule.pModule_Heff)[idx] = Hexch;
			if (do_reduction && cuModule.pModule_energy->linear_size()) (*cuModule.pModule_energy)[idx] = -(cuBReal)MUB_MU0 * M1[idx] * Hexch / (2 * M1.h.dim());
		}

		Heff1[idx] += Hexch;
	}

	if (do_reduction) reduction_sum(0, 1, &energy_, *cuModule.penergy);
}

//----------------------- UpdateField LAUNCHER

void Atom_ExchangeCUDA::UpdateField(void)
{
	if (paMeshCUDA->GetMeshType() == MESH_ATOM_CUBIC) {

		//atomistic simple cubic mesh

		if (paMeshCUDA->CurrentTimeStepSolved()) {

			ZeroEnergy();

			Atom_ExchangeCUDA_Cubic_UpdateField <<< (paMeshCUDA->n.dim() + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> (paMeshCUDA->cuaMesh, cuModule, true);
		}
		else {

			Atom_ExchangeCUDA_Cubic_UpdateField <<< (paMeshCUDA->n.dim() + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> (paMeshCUDA->cuaMesh, cuModule, false);
		}
	}
}

#endif

#endif