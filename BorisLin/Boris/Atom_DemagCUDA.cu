#include "hip/hip_runtime.h"
#include "Atom_DemagCUDA.h"

#if COMPILECUDA == 1

#if defined(MODULE_COMPILATION_DEMAG) && ATOMISTIC == 1

#include "BorisCUDALib.cuh"
#include "Atom_MeshCUDA.h"

//----------------------- Initialization

__global__ void set_Atom_DemagCUDA_pointers_kernel(
	ManagedAtom_MeshCUDA& cuaMesh, cuVEC<cuReal3>& Module_Heff)
{
	if (threadIdx.x == 0) cuaMesh.pAtom_Demag_Heff = &Module_Heff;
}

void Atom_DemagCUDA::set_Atom_DemagCUDA_pointers(void)
{
	set_Atom_DemagCUDA_pointers_kernel <<< 1, CUDATHREADS >>>
		(paMeshCUDA->cuaMesh, Module_Heff);
}

//----------------------- Auxiliary

__global__ void Atom_Demag_EvalSpeedup_SubSelf_Kernel(
	cuVEC<cuReal3>& H,
	cuVEC<cuReal3>& M, cuReal3& selfDemagCoeff)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < H.linear_size()) {

		H[idx] -= (selfDemagCoeff & M[idx]);
	}
}

//QUINTIC
__global__ void Atom_Demag_EvalSpeedup_SetExtrapField_AddSelf_Kernel(
	cuVEC<cuReal3>& H,
	cuVEC<cuReal3>& Hdemag, cuVEC<cuReal3>& Hdemag2, cuVEC<cuReal3>& Hdemag3, cuVEC<cuReal3>& Hdemag4, cuVEC<cuReal3>& Hdemag5, cuVEC<cuReal3>& Hdemag6,
	cuBReal a1, cuBReal a2, cuBReal a3, cuBReal a4, cuBReal a5, cuBReal a6,
	cuVEC<cuReal3>& M, cuReal3& selfDemagCoeff)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < H.linear_size()) {

		H[idx] = Hdemag[idx] * a1 + Hdemag2[idx] * a2 + Hdemag3[idx] * a3 + Hdemag4[idx] * a4 + Hdemag5[idx] * a5 + Hdemag6[idx] * a6 + (selfDemagCoeff & M[idx]);
	}
}

//QUARTIC
__global__ void Atom_Demag_EvalSpeedup_SetExtrapField_AddSelf_Kernel(
	cuVEC<cuReal3>& H,
	cuVEC<cuReal3>& Hdemag, cuVEC<cuReal3>& Hdemag2, cuVEC<cuReal3>& Hdemag3, cuVEC<cuReal3>& Hdemag4, cuVEC<cuReal3>& Hdemag5,
	cuBReal a1, cuBReal a2, cuBReal a3, cuBReal a4, cuBReal a5,
	cuVEC<cuReal3>& M, cuReal3& selfDemagCoeff)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < H.linear_size()) {

		H[idx] = Hdemag[idx] * a1 + Hdemag2[idx] * a2 + Hdemag3[idx] * a3 + Hdemag4[idx] * a4 + Hdemag5[idx] * a5 + (selfDemagCoeff & M[idx]);
	}
}

//CUBIC
__global__ void Atom_Demag_EvalSpeedup_SetExtrapField_AddSelf_Kernel(
	cuVEC<cuReal3>& H,
	cuVEC<cuReal3>& Hdemag, cuVEC<cuReal3>& Hdemag2, cuVEC<cuReal3>& Hdemag3, cuVEC<cuReal3>& Hdemag4,
	cuBReal a1, cuBReal a2, cuBReal a3, cuBReal a4,
	cuVEC<cuReal3>& M, cuReal3& selfDemagCoeff)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < H.linear_size()) {

		H[idx] = Hdemag[idx] * a1 + Hdemag2[idx] * a2 + Hdemag3[idx] * a3 + Hdemag4[idx] * a4 + (selfDemagCoeff & M[idx]);
	}
}

//QUADRATIC
__global__ void Atom_Demag_EvalSpeedup_SetExtrapField_AddSelf_Kernel(
	cuVEC<cuReal3>& H,
	cuVEC<cuReal3>& Hdemag, cuVEC<cuReal3>& Hdemag2, cuVEC<cuReal3>& Hdemag3,
	cuBReal a1, cuBReal a2, cuBReal a3,
	cuVEC<cuReal3>& M, cuReal3& selfDemagCoeff)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < H.linear_size()) {

		H[idx] = Hdemag[idx] * a1 + Hdemag2[idx] * a2 + Hdemag3[idx] * a3 + (selfDemagCoeff & M[idx]);
	}
}

//LINEAR
__global__ void Atom_Demag_EvalSpeedup_SetExtrapField_AddSelf_Kernel(
	cuVEC<cuReal3>& H,
	cuVEC<cuReal3>& Hdemag, cuVEC<cuReal3>& Hdemag2,
	cuBReal a1, cuBReal a2,
	cuVEC<cuReal3>& M, cuReal3& selfDemagCoeff)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < H.linear_size()) {

		H[idx] = Hdemag[idx] * a1 + Hdemag2[idx] * a2 + (selfDemagCoeff & M[idx]);
	}
}

//STEP
__global__ void Atom_Demag_EvalSpeedup_SetExtrapField_AddSelf_Kernel(
	cuVEC<cuReal3>& H,
	cuVEC<cuReal3>& Hdemag,
	cuVEC<cuReal3>& M, cuReal3& selfDemagCoeff)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < H.linear_size()) {

		H[idx] = Hdemag[idx] + (selfDemagCoeff & M[idx]);
	}
}

//----------------------- LAUNCHERS

void Atom_DemagCUDA::Atom_Demag_EvalSpeedup_SubSelf(cu_obj<cuVEC<cuReal3>>& H)
{
	Atom_Demag_EvalSpeedup_SubSelf_Kernel <<< (paMeshCUDA->n_dm.dim() + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
		(H, M, selfDemagCoeff);
}

//QUINTIC
void Atom_DemagCUDA::Atom_Demag_EvalSpeedup_SetExtrapField_AddSelf(cu_obj<cuVEC<cuReal3>>& H, cuBReal a1, cuBReal a2, cuBReal a3, cuBReal a4, cuBReal a5, cuBReal a6)
{
	Atom_Demag_EvalSpeedup_SetExtrapField_AddSelf_Kernel << < (paMeshCUDA->n_dm.dim() + CUDATHREADS) / CUDATHREADS, CUDATHREADS >> >
		(H, Hdemag, Hdemag2, Hdemag3, Hdemag4, Hdemag5, Hdemag6, a1, a2, a3, a4, a5, a6, M, selfDemagCoeff);
}

//QUARTIC
void Atom_DemagCUDA::Atom_Demag_EvalSpeedup_SetExtrapField_AddSelf(cu_obj<cuVEC<cuReal3>>& H, cuBReal a1, cuBReal a2, cuBReal a3, cuBReal a4, cuBReal a5)
{
	Atom_Demag_EvalSpeedup_SetExtrapField_AddSelf_Kernel << < (paMeshCUDA->n_dm.dim() + CUDATHREADS) / CUDATHREADS, CUDATHREADS >> >
		(H, Hdemag, Hdemag2, Hdemag3, Hdemag4, Hdemag5, a1, a2, a3, a4, a5, M, selfDemagCoeff);
}

//CUBIC
void Atom_DemagCUDA::Atom_Demag_EvalSpeedup_SetExtrapField_AddSelf(cu_obj<cuVEC<cuReal3>>& H, cuBReal a1, cuBReal a2, cuBReal a3, cuBReal a4)
{
	Atom_Demag_EvalSpeedup_SetExtrapField_AddSelf_Kernel << < (paMeshCUDA->n_dm.dim() + CUDATHREADS) / CUDATHREADS, CUDATHREADS >> >
		(H, Hdemag, Hdemag2, Hdemag3, Hdemag4, a1, a2, a3, a4, M, selfDemagCoeff);
}

//QUADRATIC
void Atom_DemagCUDA::Atom_Demag_EvalSpeedup_SetExtrapField_AddSelf(cu_obj<cuVEC<cuReal3>>& H, cuBReal a1, cuBReal a2, cuBReal a3)
{
	Atom_Demag_EvalSpeedup_SetExtrapField_AddSelf_Kernel <<< (paMeshCUDA->n_dm.dim() + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
		(H, Hdemag, Hdemag2, Hdemag3, a1, a2, a3, M, selfDemagCoeff);
}

//LINEAR
void Atom_DemagCUDA::Atom_Demag_EvalSpeedup_SetExtrapField_AddSelf(cu_obj<cuVEC<cuReal3>>& H, cuBReal a1, cuBReal a2)
{
	Atom_Demag_EvalSpeedup_SetExtrapField_AddSelf_Kernel <<< (paMeshCUDA->n_dm.dim() + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
		(H, Hdemag, Hdemag2, a1, a2, M, selfDemagCoeff);
}

//STEP
void Atom_DemagCUDA::Atom_Demag_EvalSpeedup_SetExtrapField_AddSelf(cu_obj<cuVEC<cuReal3>>& H)
{
	Atom_Demag_EvalSpeedup_SetExtrapField_AddSelf_Kernel <<< (paMeshCUDA->n_dm.dim() + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
		(H, Hdemag, M, selfDemagCoeff);
}

#endif

#endif